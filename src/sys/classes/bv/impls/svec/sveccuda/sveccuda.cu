/*
   BV implemented as a single Vec (CUDA version)

   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
   SLEPc - Scalable Library for Eigenvalue Problem Computations
   Copyright (c) 2002-2015, Universitat Politecnica de Valencia, Spain

   This file is part of SLEPc.

   SLEPc is free software: you can redistribute it and/or modify it under  the
   terms of version 3 of the GNU Lesser General Public License as published by
   the Free Software Foundation.

   SLEPc  is  distributed in the hope that it will be useful, but WITHOUT  ANY
   WARRANTY;  without even the implied warranty of MERCHANTABILITY or  FITNESS
   FOR  A  PARTICULAR PURPOSE. See the GNU Lesser General Public  License  for
   more details.

   You  should have received a copy of the GNU Lesser General  Public  License
   along with SLEPc. If not, see <http://www.gnu.org/licenses/>.
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
*/

#include <slepc/private/bvimpl.h>
#include "../svecimpl.h"
#include <petsccuda.h>
#include <hipblas.h>

/* complex single */
#if defined(PETSC_USE_COMPLEX)
#if defined(PETSC_USE_REAL_SINGLE)
#define cublasXgemm(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipblasCgemm((a),(b),(c),(d),(e),(f),(hipComplex*)(g),(hipComplex*)(h),(i),(hipComplex*)(j),(k),(hipComplex*)(l),(hipComplex*)(m),(n))
#define cublasXgemv(a,b,c,d,e,f,g,h,i,j,k,l) hipblasCgemv((a),(b),(c),(d),(hipComplex*)(e),(hipComplex*)(f),(g),(hipComplex*)(h),(i),(hipComplex*)(j),(hipComplex*)(k),(l))
#else /* complex double */
#define cublasXgemm(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipblasZgemm((a),(b),(c),(d),(e),(f),(hipDoubleComplex*)(g),(hipDoubleComplex*)(h),(i),(hipDoubleComplex*)(j),(k),(hipDoubleComplex*)(l),(hipDoubleComplex*)(m),(n))
#define cublasXgemv(a,b,c,d,e,f,g,h,i,j,k,l) hipblasZgemv((a),(b),(c),(d),(hipDoubleComplex*)(e),(hipDoubleComplex*)(f),(g),(hipDoubleComplex*)(h),(i),(hipDoubleComplex*)(j),(hipDoubleComplex*)(k),(l))
#endif
#else /* real single */
#if defined(PETSC_USE_REAL_SINGLE)
#define cublasXgemm hipblasSgemm
#define cublasXgemv hipblasSgemv
#else /* real double */
#define cublasXgemm hipblasDgemm
#define cublasXgemv hipblasDgemv
#endif
#endif

#undef __FUNCT__
#define __FUNCT__ "BVMult_Svec_CUDA"
PetscErrorCode BVMult_Svec_CUDA(BV Y,PetscScalar alpha,PetscScalar beta,BV X,Mat Q)
{
  PetscErrorCode ierr;
  BV_SVEC        *y = (BV_SVEC*)Y->data,*x = (BV_SVEC*)X->data;
  PetscScalar    *px,*py,*q,*d_q;
  PetscInt       ldq,mq;
  hipblasStatus_t cberr;
  hipError_t    err;

  PetscFunctionBegin;
  if (!Y->n) PetscFunctionReturn(0);
  ierr = MatGetSize(Q,&ldq,&mq);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(x->v,&px);CHKERRQ(ierr);
  if (beta==0.0) {
    ierr = VecCUDAGetArrayWrite(y->v,&py);CHKERRQ(ierr);
  } else {
    ierr = VecCUDAGetArrayReadWrite(y->v,&py);CHKERRQ(ierr);
  }
  ierr = MatDenseGetArray(Q,&q);CHKERRQ(ierr);
  err = hipMalloc((void**)&d_q,ldq*mq*sizeof(PetscScalar*));CHKERRCUDA(err);
  err = hipMemcpy(d_q,q,ldq*mq*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);

  cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_N,Y->n,Y->k-Y->l,X->k-X->l,(const PetscScalar*)&alpha,px+(X->nc+X->l)*X->n,Y->n,d_q+Y->l*ldq+X->l,ldq,(const PetscScalar*)&beta,py+(Y->nc+Y->l)*Y->n,Y->n);CHKERRCUBLAS(cberr);

  ierr = MatDenseRestoreArray(Q,&q);CHKERRQ(ierr);
  err = hipFree(d_q);CHKERRCUDA(err);
  ierr = VecCUDARestoreArrayRead(x->v,&px);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(y->v,&py);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

